#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
//TODO: BIG INT is needed
__global__ void decyrption(int *M, int *C, int *d, int *N){
    
    int i = threadIdx.x;
    M[i] = pow(M[i], d[i]);
    M[i] = M[i] % N[i];

}

int main(){
    int C[4] = {541, 795, 1479, 2753};
    int M[4];
    int d[4] = {1019, 1019, 1019, 1019};
    int N[4] = {3337, 3337, 3337, 3337};

    int *C_GPU, *M_GPU, *d_GPU, *N_GPU;
    int size = 4 * sizeof(int);

    hipMalloc((void **)&C_GPU, size);
    hipMalloc((void **)&M_GPU, size);
    hipMalloc((void **)&d_GPU, size);
    hipMalloc((void **)&N_GPU, size);

    hipMemcpy(C_GPU, C, size, hipMemcpyHostToDevice);
    hipMemcpy(d_GPU, d, size, hipMemcpyHostToDevice);
    hipMemcpy(N_GPU, N, size, hipMemcpyHostToDevice);

    decyrption<<<1, 4>>>(M_GPU, C_GPU, d_GPU, N_GPU);

    hipMemcpy(M, M_GPU, size, hipMemcpyDeviceToHost);
    hipFree(M_GPU);
    hipFree(C_GPU);
    hipFree(d_GPU);
    hipFree(N_GPU);

    int i;
    for (i = 0; i < 4; i++){
        printf("The result is %d, %d\n", M[i], C[i]);
    }
}
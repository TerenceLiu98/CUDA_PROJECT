/* 
Basically, the substraction are the same to the `add.cu`
But be care, DO NOT use `+` to do the substraction
please use `-`! (JUST FOR FUN) :)
*/

#include <stdio.h>
#include "hip/hip_runtime.h"

// CUDA Kernel Function 

__global__ void add(int *a, int *b, int *c){
    int i = threadIdx.x;
    c[i] = b[i] + a[i];
}

// main Function

int main(){
    // define A, B, and C
    // These are three array and we will do A + B = C
    int A[5] = {1, 2, 3, 4, 5};
    int B[5] = {7, 8, 10, 18, 20};
    int C[5];
    int C_check[5];

    // define arrays for A, B, and C 
    // and copy these memory from host memory to device memory
    int *A_gpu;
    int *B_gpu;
    int *C_gpu;

    int size = 5 * sizeof(int); 

    // allocate memory for A, B, and C
    hipMalloc((void **)&A_gpu, size);
    hipMalloc((void **)&B_gpu, size);
    hipMalloc((void **)&C_gpu, size);

    // copy the memory
    hipMemcpy(A_gpu, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, size, hipMemcpyHostToDevice);

    add<<<1, 5>>>(A_gpu, B_gpu, C_gpu);

    // copy memory from device to host 
    // since the result is stored in C_gpu
    // we just need this memory
    hipMemcpy(C, C_gpu, size, hipMemcpyDeviceToHost);



    int i;
    
    for (i = 0; i < 5; i++){
        C_check[i] = A[i] - B[i];
    }
    
    
    for (i = 0; i < 5; i++){
        printf("The Sum is %d\n", C[i] == C_check[i]);
    }

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);

    return 0;
}
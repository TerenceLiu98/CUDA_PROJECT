#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void normal_devision(int *a, int *b, int *c){
    // for the small case, we only need thread, no block and grid
    int i = threadIdx.x;
    c[i] = a[i] / b[i];
    // printf is not allowed in kernel function
    // printf("%d", c[i])
}

int main(){
    int a[3] = {1, 2, 5};
    int b[3] = {2, 4, 6};
    int c[3]; // int array
    int c_check[3];
    int *A_gpu, *B_gpu, *C_gpu; // pointers
    int size = 5 * sizeof(int);

    // allocate memory for A, B, and C
    hipMalloc((void **)&A_gpu, size);
    hipMalloc((void **)&B_gpu, size);
    hipMalloc((void **)&C_gpu, size);

    // copy the memory
    hipMemcpy(A_gpu, a, size, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, b, size, hipMemcpyHostToDevice);

    division<<<1, 3>>>(A_gpu, B_gpu, C_gpu);

    // copy memory from device to host 
    // since the result is stored in C_gpu
    // we just need this memory
    hipMemcpy(c, C_gpu, size, hipMemcpyDeviceToHost);



    int i;
    
    for (i = 0; i < 3; i++){
        c_check[i] = a[i] / b[i];
    }
    
    
    for (i = 0; i < 3; i++){
        printf("The Sum is %d\n", c[i] == c_check[i]);
    }

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);

    return 0;
}


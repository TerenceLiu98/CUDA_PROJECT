#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void modular(int *a, int *b, int *c){
    // for the small case, we only need thread, no block and grid
    int i = threadIdx.x;
    c[i] = a[i] % b[i];
    // printf is not allowed in kernel function
    // printf("%d", c[i])
}

__global__ void exponentiation(int *a, int *b, int *d){
    // for the small case, we only need thread, no block and grid
    int i = threadIdx.x;
    d[i] = pow(a[i], b[i]);
}

int main(){
    int a[3] = {1, 2, 5}, b[3] = {2, 4, 6};
    int c[3], d[3], i;
    int c_check[3],  d_exponentiation[3];
    int *A_gpu, *B_gpu, *C_gpu, *D_gpu; // pointers
    int size = 5 * sizeof(int);

    // allocate memory for A, B, and C
    hipMalloc((void **)&A_gpu, size);
    hipMalloc((void **)&B_gpu, size);
    hipMalloc((void **)&C_gpu, size);
    hipMalloc((void **)&D_gpu, size);

    // copy the memory
    hipMemcpy(A_gpu, a, size, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, b, size, hipMemcpyHostToDevice);

    modular<<<1, 3>>>(A_gpu, B_gpu, C_gpu);
    modular<<<1, 3>>>(A_gpu, B_gpu, D_gpu);

    // copy memory from device to host 
    // since the result is stored in C_gpu
    // we just need this memory
    hipMemcpy(c, C_gpu, size, hipMemcpyDeviceToHost);
    hipMemcpy(d, D_gpu, size, hipMemcpyDeviceToHost);

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
    hipFree(D_gpu);
    
    for (i = 0; i < 3; i++){
        c_check[i] = a[i] % b[i];
        printf("The GPU Version %d, The CPU Version is %d\n", c[i], c_check[i]);
    }  
    
//TODO: why the CPU version in exponentiation is wrong ? WHY ?
    for (i = 0; i < 3; i++){
        d_exponentiation[i] = pow(a[i], b[i]);
        printf("The GPU Version %d, The CPU Version is %d\n", d[i], d_exponentiation[i]);
    }

    return 0;
}

